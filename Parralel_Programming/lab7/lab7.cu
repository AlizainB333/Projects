#include "hip/hip_runtime.h"
#include <wb.h>

#define HISTOGRAM_LENGTH 256

//@@ insert code here


__global__ void convertFloatChar(float* inputImage, unsigned char* outputImage, int imgLenght) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx < imgLenght) {
    outputImage[idx] = (unsigned char) (255 * inputImage[idx]);
  }
}

__global__ void convertCharFloat(unsigned char* inputImage, float* outputImage, int imgLenght) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx < imgLenght) {
    outputImage[idx] = (float) (inputImage[idx] / 255.0);
  }
}

__global__ void RGBToGrayScale(unsigned char* rgbImage, unsigned char* grayscaleImage, int imgLenght) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int r,g,b;

  if(idx < imgLenght) {
    r = rgbImage[3*idx];
		g = rgbImage[3*idx + 1];
		b = rgbImage[3*idx + 2];
		grayscaleImage[idx] = (unsigned char) (0.21*r + 0.71*g + 0.07*b);
  }
}

__global__ void computeHistogram(unsigned char* buffer, unsigned int* histo, int size) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  
  __shared__ unsigned int histo_private[256];
  if (threadIdx.x < 256) {
    histo_private[threadIdx.x] = 0;
  }
  __syncthreads();

  if (i < size) {
    atomicAdd(&(histo_private[buffer[i]]), 1);
  }
  __syncthreads();

  if (threadIdx.x < 256) {
    atomicAdd(&(histo[threadIdx.x]),histo_private[threadIdx.x]);
  }
}

__global__ void histoEqualization(float* cdf, unsigned char* outputImg, int length) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;

  if(i < length) {
    float val = 255*(cdf[outputImg[i]] - cdf[0])/(1.0 - cdf[0]);
    float clamp = min(max(val,0.0f),255.0f);
    outputImg[i] = clamp;
  }
}

int main(int argc, char **argv) {
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  int imageChannels;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  const char *inputImageFile;

  // Device Init 
  float *deviceInputImageData;
  float *deviceOutputImageData;
  unsigned char *deviceUnsignedImageData;
  unsigned char *deviceRGBGrayscaleData;
  unsigned int *deviceHistogram;
  unsigned int *hostHistogram;
  float *deviceCdf;
  float *hostCdf;


  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  //Import data and create memory on host
  inputImage = wbImport(inputImageFile);
  imageWidth = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);
  outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);

  hostInputImageData = inputImage->data;

  //@@ insert code here
  int size = imageWidth*imageHeight*imageChannels;
  int grayscaleSize = imageWidth*imageHeight;
  int dimBlockSize = ceil((1.0*size) / HISTOGRAM_LENGTH);

  hipMalloc((void**) &deviceInputImageData, sizeof(float) * size);
  hipMalloc((void**) &deviceOutputImageData, sizeof(float) * size);
  hipMalloc((void**) &deviceUnsignedImageData, sizeof(unsigned char) * size);
  hipMalloc((void**) &deviceRGBGrayscaleData, sizeof(unsigned char) * grayscaleSize);
  hipMalloc((void**) &hostHistogram, sizeof(unsigned int) * HISTOGRAM_LENGTH);
  hipMalloc((void**) &hostCdf, sizeof(float) * HISTOGRAM_LENGTH);
  deviceHistogram = (unsigned int *) malloc(sizeof(unsigned int) * HISTOGRAM_LENGTH);
  deviceCdf = (float *) malloc(sizeof(float) * HISTOGRAM_LENGTH);


  hipMemcpy(deviceInputImageData, hostInputImageData, sizeof(float) * size, hipMemcpyHostToDevice);

  dim3 dimBlock(HISTOGRAM_LENGTH, 1, 1);
  dim3 dimGrid((dimBlockSize), 1, 1);

  // COnvert Img to grayscale then unschar->float first

  convertFloatChar<<<dimGrid, dimBlock>>>(deviceInputImageData, deviceUnsignedImageData, size);
  RGBToGrayScale<<<dimGrid, dimBlock>>>(deviceUnsignedImageData, deviceRGBGrayscaleData, grayscaleSize);
  computeHistogram<<<dimGrid, dimBlock>>>(deviceRGBGrayscaleData, hostHistogram, grayscaleSize);
  // Check if cpy right 
  hipMemcpy(deviceHistogram, hostHistogram, sizeof(unsigned int) * HISTOGRAM_LENGTH, hipMemcpyDeviceToHost);

  //Include cpu code
  deviceCdf[0] = deviceHistogram[0] / (1.0 * grayscaleSize);
  for(int i = 0; i < 256; i++)
  {
    deviceCdf[i] = deviceCdf[i - 1] + deviceHistogram[i] / (1.0 * grayscaleSize);
  }
  // compute histo then back to usnigned to restore value
  hipMemcpy(hostCdf, deviceCdf, sizeof(float) * HISTOGRAM_LENGTH, hipMemcpyHostToDevice);
  histoEqualization<<<dimGrid, dimBlock>>>(hostCdf, deviceUnsignedImageData, size);
  // printf(dimGrid)
  convertCharFloat<<<dimGrid, dimBlock>>>(deviceUnsignedImageData, deviceOutputImageData, size);

  hostOutputImageData = outputImage->data;
  hipMemcpy(hostOutputImageData, deviceOutputImageData, sizeof(float) * size, hipMemcpyDeviceToHost);

  wbSolution(args, outputImage);

  //@@ insert code here


// Free Host 
  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);
  hipFree(deviceUnsignedImageData);
  hipFree(deviceRGBGrayscaleData);
  hipFree(hostHistogram);
  hipFree(hostCdf);

//Free Malloc 
  free(deviceHistogram);
  free(deviceCdf);

  return 0;
}
