#include "hip/hip_runtime.h"
// LAB 2 FA24

#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)


// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns)
{
  //@@ Implement matrix multiplication kernel here
  // Mat formaula Index*Dim*Idx

    int matrixRowDim = blockIdx.y * blockDim.y + threadIdx.y;  // Row of matrix kernel
    int matrixColDim = blockIdx.x * blockDim.x + threadIdx.x;  // Colomns Idx of matrix
    float matrixInitialVal;

    // Check first if row dim is not exceeding num rows and
    // colomns not exceeding max col dim
    if (matrixRowDim < numCRows && matrixColDim < numCColumns) {
      // Reset Value for every calculation
      matrixInitialVal = 0.0;
      
      for (int i = 0; i < numAColumns; ++i) {
        // Formula Row*maxCol*Index + Index*col+COlDims
        matrixInitialVal += A[(matrixRowDim * numAColumns) + i] * B[(numCColumns * i) + matrixColDim];  // Cal new val
      }
      C[matrixRowDim * numCColumns + matrixColDim] = matrixInitialVal;  // Update output matrix with new calculated value
    }
}


int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)




  args = wbArg_read(argc, argv);

  //@@ Importing data and creating memory on host
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);



  //@@ Set numCRows and numCColumns
  // Initlize Device Name
  float *deviceA; 
  float *deviceB; 
  float *deviceC;
  // Set from init value given
  numCColumns = numBColumns;
  numCRows = numARows;

  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));


  //@@ Allocate GPU memory here
  // Source,Size
  hipMalloc((void **)&deviceA, numARows * numAColumns * sizeof(float));
  hipMalloc((void **)&deviceB, numBRows * numBColumns * sizeof(float));
  hipMalloc((void **)&deviceC, numCRows * numCColumns * sizeof(float));

  //@@ Copy memory to the GPU here
  // Destination,Source,Size
  hipMemcpy(deviceA, hostA, (numARows*numAColumns * sizeof(float)), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, (numBRows*numBColumns * sizeof(float)), hipMemcpyHostToDevice);

  //@@ Initialize the grid and block dimensions here
  dim3 dimBlock(16, 16); // Thread Block Size
  // Set Grid Dim from formula lecture
  dim3 dimGrid((numCRows + dimBlock.x - 1) / dimBlock.x, (numCColumns + dimBlock.y - 1) / dimBlock.y);

  // Entire Block Coverage // Init Grid Slide 6
  matrixMultiply<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCColumns, numCRows);

  //@@ Launch the GPU Kernel here
  hipDeviceSynchronize();  // From Lec 4
  
  //@@ Copy the GPU memory back to the CPU here
  // Destination, Source, Size
  hipMemcpy(hostC, deviceC, (numCRows*numCColumns*sizeof(float)), hipMemcpyDeviceToHost);

  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  wbSolution(args, hostC, numCRows, numCColumns);

  // Both Inout Hosts muust be freed
  free(hostA);
  free(hostB);
  free(hostC);
  //@@Free the hostC matrix

  return 0;
}

