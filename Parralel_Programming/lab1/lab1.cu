#include "hip/hip_runtime.h"
// LAB 1
#include <wb.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
  int i = blockIdx.x * blockDim.x + threadIdx.x ;
  if(i < len) out[i] = in1[i] + in2[i];
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;

  args = wbArg_read(argc, argv);
  //@@ Importing data and creating memory on host
  hostInput1 =
      (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostInput2 =
      (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  wbLog(TRACE, "The input length is ", inputLength);

  //@@ Allocate GPU memory here
  float *vec1,*vec2,*outVec;
  int size = inputLength * sizeof(float);
  hipMalloc((void **) &vec1, size);
  hipMemcpy(vec1, hostInput1, size, hipMemcpyHostToDevice);
  hipMalloc((void **) &vec2, size);
  hipMemcpy(vec2, hostInput2, size, hipMemcpyHostToDevice);

  //@@ Copy memory to the GPU here
  hipMalloc((void **) &outVec, size);

  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(inputLength/256, 1, 1);
  if (0 != (inputLength % 256)) { DimGrid.x++; }
  dim3 DimBlock(256, 1, 1);
  vecAdd<<<DimGrid,DimBlock>>>(vec1, vec2, outVec, inputLength);

  //@@ Launch the GPU Kernel here to perform CUDA computation
  hipDeviceSynchronize();

  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, outVec, size, hipMemcpyDeviceToHost);


  //@@ Free the GPU memory here
  hipFree(vec1); hipFree(vec2); hipFree(outVec);

  wbSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}
