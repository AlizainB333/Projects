#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16
#define BLOCK_SIZE 256



 __global__ void KenelFusedConvulution(const int Channel, const int K, const int Height, const int Width, const int Width_out, const int Height_out, const int Map_out,
                                       const float* inputTile, const float* tileMask, float* outputTile) {
                                            
    __shared__ float tileMatA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tileMatB[TILE_WIDTH][TILE_WIDTH];

    #define in_4d(i3, i2, i1, i0)   inputTile[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define mask_4d(i3, i2, i1, i0) tileMask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]
    #define out_4d(i3, i2, i1, i0)  outputTile[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]

    int batch_size = blockIdx.z;

    int tx = threadIdx.x, ty = threadIdx.y;
    int row = blockIdx.y * TILE_WIDTH + ty;
    int column = blockIdx.x * TILE_WIDTH + tx;
    int numMatAColumns = Channel*K*K; 

    float acc = 0.0;

    int num_iterations = ceil(numMatAColumns/(1.0*TILE_WIDTH));

    for (int i = 0; i < num_iterations; i++) {
        int temp_col = i*TILE_WIDTH + tx, temp_row = i*TILE_WIDTH + ty;
        tileMatA[ty][tx] = 0;
        tileMatB[ty][tx] = 0;

        int W_c = temp_col/(K*K);
        int W_h = (temp_col%(K*K))/K, W_w = (temp_col%(K*K))%K;

        if (temp_col < numMatAColumns && row < Map_out) {
            tileMatA[ty][tx] = mask_4d(row, W_c, W_h, W_w);
        }
        else {
            tileMatA[ty][tx] = 0;
        }

        int X_c = temp_row/(K*K);
        int X_p = temp_row%(K*K)/K , X_q = (temp_row%(K*K))%K;
        int X_h = column/Width_out, X_w = column%Width_out;

        if (temp_row < numMatAColumns && column < Height_out*Width_out) {
            tileMatB[ty][tx] = in_4d(batch_size, X_c, X_h + X_p, X_w + X_q);
        }
        else {
            tileMatB[ty][tx] = 0; 
        }

        __syncthreads();

        for (int q = 0; q < TILE_WIDTH; q++) {
            acc += tileMatA[ty][q] * tileMatB[q][tx];
            __syncthreads();
        }
    }

    int Y_h = column / Width_out, Y_w = column % Width_out;

    if (row < Map_out && column < Width_out*Height_out) {
        out_4d(batch_size, row, Y_h, Y_w) = acc;
    }

    #undef in_4d
    #undef mask_4d
    #undef out_4d
 }

__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

    // Assigning to global variable access in different streams


    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    hipMalloc((void**) device_input_ptr, Batch * Channel * Height * Width * sizeof(float));
    hipMalloc((void**) device_output_ptr, Batch * Map_out * Height_out * Width_out * sizeof(float));
    hipMalloc((void**) device_mask_ptr, Map_out * Channel * K * K * sizeof(float));

    hipMemcpy(*device_input_ptr, host_input, Batch * Channel * Height * Width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, Map_out * Channel * K * K * sizeof(float), hipMemcpyHostToDevice);

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    // TODO: Set the kernel dimensions and call the fused kernel
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 DimGrid(ceil(Width_out*Height_out / float(TILE_WIDTH)), ceil(Map_out / float(TILE_WIDTH)), Batch);

    KenelFusedConvulution<<<DimGrid,DimBlock>>> (Channel, K, Width, Height, Width_out, Height_out, Map_out,
                                        device_input,  device_mask, device_output);

}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Copy the output back to host
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    hipMemcpy(host_output, device_output, Height_out * Width_out * Batch * Map_out * sizeof(float), hipMemcpyDeviceToHost);

    // TODO: Free device memory
    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask);

}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}