#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#include <mma.h>
using namespace nvcuda;

#define TILE_WIDTH 16
#define BLOCK_SIZE 256



 __global__ void KenelFusedConvulution(const int Channel, const int K, const int Height, const int Width, const int Width_out, const int Height_out, const int Map_out,
                                       const float* inputTile, const float* tileMask, float* outputTile) {
    
    __shared__ half tileA[TILE_WIDTH * TILE_WIDTH];
    __shared__ half tileB[TILE_WIDTH * TILE_WIDTH];
    __shared__ float tileC[TILE_WIDTH * TILE_WIDTH];

    #define in_4d(i3, i2, i1, i0)   inputTile[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define mask_4d(i3, i2, i1, i0) tileMask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]
    #define out_4d(i3, i2, i1, i0)  outputTile[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]

    int batch_size = blockIdx.z;
    int tx = threadIdx.x, ty = threadIdx.y;
    int row = blockIdx.y * TILE_WIDTH + ty;
    int column = blockIdx.x * TILE_WIDTH + tx;
    int numMatAColumns = Channel * K * K; 

    wmma::fragment<wmma::matrix_a, TILE_WIDTH, TILE_WIDTH, TILE_WIDTH, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, TILE_WIDTH, TILE_WIDTH, TILE_WIDTH, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, TILE_WIDTH, TILE_WIDTH, TILE_WIDTH, float> acc_frag;

    wmma::fill_fragment(acc_frag, 0.0f);

    int num_iterations = (numMatAColumns + TILE_WIDTH - 1) / TILE_WIDTH;

    for (int i = 0; i < num_iterations; i++) {
        int temp_col = i * TILE_WIDTH + tx;
        int temp_row = i * TILE_WIDTH + ty;

        // Initialize shared memory tiles
        if (temp_col < numMatAColumns && row < Map_out) {
            int W_c = temp_col / (K * K);
            int W_h = (temp_col % (K * K)) / K;
            int W_w = (temp_col % (K * K)) % K;
            tileA[ty * TILE_WIDTH + tx] = __float2half(mask_4d(row, W_c, W_h, W_w));
        } else {
            tileA[ty * TILE_WIDTH + tx] = __float2half(0.0f);
        }

        if (temp_row < numMatAColumns && column < Height_out * Width_out) {
            int X_c = temp_row / (K * K);
            int X_p = (temp_row % (K * K)) / K;
            int X_q = (temp_row % (K * K)) % K;
            int X_h = column / Width_out;
            int X_w = column % Width_out;
            tileB[ty * TILE_WIDTH + tx] = __float2half(in_4d(batch_size, X_c, X_h + X_p, X_w + X_q));
        } else {
            tileB[ty * TILE_WIDTH + tx] = __float2half(0.0f);
        }

        __syncthreads();

        // Perform matrix multiplication using Tensor Cores
        wmma::load_matrix_sync(a_frag, tileA, TILE_WIDTH);
        wmma::load_matrix_sync(b_frag, tileB, TILE_WIDTH);
        wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);

        __syncthreads();
    }

    wmma::store_matrix_sync(tileC, acc_frag, TILE_WIDTH, wmma::mem_row_major);

    int Y_h = column / Width_out;
    int Y_w = column % Width_out;

    if (row < Map_out && column < Width_out * Height_out) {
        out_4d(batch_size, row, Y_h, Y_w) = tileC[ty * TILE_WIDTH + tx];
    }

    #undef in_4d
    #undef mask_4d
    #undef out_4d
 }

__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

    // Assigning to global variable access in different streams


    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    hipMalloc((void**) device_input_ptr, Batch * Channel * Height * Width * sizeof(float));
    hipMalloc((void**) device_output_ptr, Batch * Map_out * Height_out * Width_out * sizeof(float));
    hipMalloc((void**) device_mask_ptr, Map_out * Channel * K * K * sizeof(float));

    hipMemcpy(*device_input_ptr, host_input, Batch * Channel * Height * Width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, Map_out * Channel * K * K * sizeof(float), hipMemcpyHostToDevice);

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    // TODO: Set the kernel dimensions and call the fused kernel
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 DimGrid(ceil(Width_out*Height_out / float(TILE_WIDTH)), ceil(Map_out / float(TILE_WIDTH)), Batch);

    KenelFusedConvulution<<<DimGrid,DimBlock>>> (Channel, K, Width, Height, Width_out, Height_out, Map_out,
                                        device_input,  device_mask, device_output);

}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Copy the output back to host
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    hipMemcpy(host_output, device_output, Height_out * Width_out * Batch * Map_out * sizeof(float), hipMemcpyDeviceToHost);

    // TODO: Free device memory
    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask);

}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}