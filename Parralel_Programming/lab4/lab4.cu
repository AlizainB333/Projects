#include "hip/hip_runtime.h"
#include <wb.h>


#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ Define any useful program-wide constants here
// Strategy Three
#define current_tile_arr_WIDTH_DIM 5
#define MASK_WIDTH_DIM 3
//@@ Define constant memory for device kernel here

__constant__ float M_dim_tile[MASK_WIDTH_DIM][MASK_WIDTH_DIM][MASK_WIDTH_DIM];

__global__ void conv3d(float *input, float *output, const int z_size,
                       const int y_size, const int x_size) {
  //@@ Insert kernel code here
  // Dim for shared current_tile_arr
  __shared__ float current_tile_arr[current_tile_arr_WIDTH_DIM + 2][current_tile_arr_WIDTH_DIM + 2][current_tile_arr_WIDTH_DIM + 2];

  int thread_x = threadIdx.x;
  int thread_y = threadIdx.y;
  int thread_z = threadIdx.z;

  int row_inp = blockIdx.y * current_tile_arr_WIDTH_DIM + thread_y;
  int col_inp = blockIdx.x * current_tile_arr_WIDTH_DIM + thread_x;
  int z_inp =   blockIdx.z * current_tile_arr_WIDTH_DIM + thread_z;

  int row_i = row_inp - (MASK_WIDTH_DIM - 1) / 2; 
  int col_i = col_inp - (MASK_WIDTH_DIM - 1) / 2; 
  int z_i = z_inp - (MASK_WIDTH_DIM - 1)/2;



  float Pvalue = 0.0f;
  if ((row_i >= 0) && (row_i < y_size) && (col_i >= 0) && (col_i < x_size) && (z_i >= 0) && (z_i < z_size)) {
    current_tile_arr[thread_z][thread_y][thread_x] = input[z_i * x_size * y_size + row_i * x_size + col_i];
    } 
    else{
      current_tile_arr[thread_z][thread_y][thread_x] = 0.0f;
      }
    __syncthreads (); // wait for current_tile_arr

    if ((thread_y < current_tile_arr_WIDTH_DIM) && (thread_x < current_tile_arr_WIDTH_DIM) && (thread_z < current_tile_arr_WIDTH_DIM)) {
      // Outer kernel loop for tile 0
        for(int i = 0; i < MASK_WIDTH_DIM; i++) {
          // GO through inner dimension tile width 
          for(int j = 0; j < MASK_WIDTH_DIM; j++) {
            for(int k = 0; k < MASK_WIDTH_DIM; k++){
              Pvalue += M_dim_tile[i][j][k] * current_tile_arr[i+thread_z][j+thread_y][k+thread_x];
              }
          }
        }
        if((row_inp < y_size) && (col_inp < x_size) && (z_inp < z_size)){
          output[z_inp * x_size * y_size + row_inp * x_size + col_inp] = Pvalue;
        }
    }
  }

                       

int main(int argc, char *argv[]) {
  wbArg_t args;
  int z_size;
  int y_size;
  int x_size;
  int tile_size;
  int tile_n_size;
  int tile_x_size;
  int inputLength, kernelLength;
  float *hostInput;
  float *hostKernel;
  float *hostOutput;
  //@@ Initial deviceInput and deviceOutput here.
  float *deviceInput;
  float *deviceOutput;


  args = wbArg_read(argc, argv);

  // Import data

  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostKernel = (float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  // First three elements are the input dimensions
  z_size = hostInput[0];
  y_size = hostInput[1];
  x_size = hostInput[2];
  wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
  assert(z_size * y_size * x_size == inputLength - 3);
  assert(kernelLength == 27);


  //@@ Allocate GPU memory here
  // Recall that inputLength is 3 elements longer than the input data
  // because the first  three elements were the dimensions
  
  hipMalloc((void **) &deviceInput, (inputLength - 3) * sizeof(float));
  hipMalloc((void **) &deviceOutput, (inputLength - 3) * sizeof(float));
  


  //@@ Copy input and kernel to GPU here
  // Recall that the first three elements of hostInput are dimensions and
  // do
  // not need to be copied to the gpu
  hipMemcpy(deviceInput, hostInput + 3, (inputLength - 3) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(M_dim_tile), hostKernel, kernelLength*sizeof(float), 0, hipMemcpyHostToDevice);

  dim3 dimGrid(ceil((1.0 * x_size)/(1.0*current_tile_arr_WIDTH_DIM)), ceil((1.0 * y_size)/(1.0*current_tile_arr_WIDTH_DIM)), ceil((1.0 * z_size)/(1.0*current_tile_arr_WIDTH_DIM)));
  dim3 dimBlock(current_tile_arr_WIDTH_DIM + MASK_WIDTH_DIM - 1, current_tile_arr_WIDTH_DIM + MASK_WIDTH_DIM - 1, current_tile_arr_WIDTH_DIM + MASK_WIDTH_DIM - 1);
  // dim3 dimBlock(current_tile_arr_WIDTH_DIM + tileWIdth, current_tile_arr_WIDTH_DIM + MASK_WIDTH_DIM - 1, current_tile_arr_WIDTH_DIM + MASK_WIDTH_DIM - 1);

  // Dim OFF by 1 Fix
  // dim3 dimBlock(current_tile_arr_WIDTH_DIM + MASK_WIDTH_DIM, current_tile_arr_WIDTH_DIM + MASK_WIDTH_DIM - 1, current_tile_arr_WIDTH_DIM + MASK_WIDTH_DIM - 1);
  dim3 dimBlock(current_tile_arr_WIDTH_DIM + MASK_WIDTH_DIM - 1, current_tile_arr_WIDTH_DIM + MASK_WIDTH_DIM - 1, current_tile_arr_WIDTH_DIM + MASK_WIDTH_DIM - 1);

  // Grid Dim


  //@@ Launch the GPU kernel here

  conv3d<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, z_size, y_size, x_size);
  hipDeviceSynchronize();



  //@@ Copy the device memory back to the host here
  // Recall that the first three elements of the output are the dimensions
  // and should not be set here (they are set below)


  hipMemcpy(hostOutput + 3, deviceOutput, (inputLength - 3) * sizeof(float), hipMemcpyDeviceToHost);
  

  // Set the output dimensions for correctness checking
  hostOutput[0] = z_size;
  hostOutput[1] = y_size;
  hostOutput[2] = x_size;
  wbSolution(args, hostOutput, inputLength);

  //@@ Free device memory

  hipFree(deviceInput);
  hipFree(deviceOutput);

  // Free host memory
  free(hostInput);
  free(hostOutput);
  return 0;
}