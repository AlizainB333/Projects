#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define TILE_WIDTH 16

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  //@@ You have to use shared memory for this MP
  __shared__ float subTileA[TILE_WIDTH][TILE_WIDTH];
  __shared__ float subTileB[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;

  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;
  float Pvalue = 0;
  
  for (int i = 0; i < (ceil(1.0*numAColumns/TILE_WIDTH)); i++) {
  // Collaborative loading of M and N tiles into shared memory

    if(Row < numARows && (i*TILE_WIDTH+tx) < numAColumns) {
      subTileA[ty][tx] = A[Row*numAColumns + i*TILE_WIDTH+tx];
    } else {
      subTileA[ty][tx] = 0;
    }

    // Bound Checking
    if((i*TILE_WIDTH+ty) < numBRows && Col < numBColumns) {
      subTileB[ty][tx] = B[ (i*TILE_WIDTH+ty) * numBColumns+Col];
    } else {
      subTileB[ty][tx] = 0;
    }

    __syncthreads();

    for (int k = 0; k < TILE_WIDTH; k++) {
      Pvalue += subTileA[ty][k] * subTileB[k][tx];
    }

    __syncthreads();
  }
  // Bound Checking
  if(Row < numCRows && Col < numCColumns) {
    C[Row*numCColumns+Col] = Pvalue;
  }
  
  
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix

  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  //@@ Importing data and creating memory on host
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);

  //@@ Set numCRows and numCColumns
  float *deviceA; 
  float *deviceB; 
  float *deviceC;

  numCColumns = numBColumns;
  numCRows = numARows;


  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));
  
  //@@ Allocate GPU memory here
  hipMalloc((void **)&deviceA, numARows * numAColumns * sizeof(float));
  hipMalloc((void **)&deviceB, numBRows * numBColumns * sizeof(float));
  hipMalloc((void **)&deviceC, numCRows * numCColumns * sizeof(float));

  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, (numARows*numAColumns * sizeof(float)), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, (numBRows*numBColumns * sizeof(float)), hipMemcpyHostToDevice);

  //@@ Initialize the grid and block dimensions here
  dim3 dimBlock(16, 16, 1); // Thread Block Size
  // Set Grid Dim from formula lecture
  // dim3 dimGrid(ceil(1.0 * numCColumns + dimBlock.x - 1) / dimBlock.x, (numCRows + dimBlock.y - 1) / dimBlock.y, 1);
  dim3 dimGrid(ceil((1.0 * numCColumns) / 16), ceil((1.0 * numCRows) / 16), 1);
  matrixMultiplyShared<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);


  //@@ Launch the GPU Kernel here
  hipDeviceSynchronize();

  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, (numCRows*numCColumns*sizeof(float)), hipMemcpyDeviceToHost);

  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);

  //@@ Free the hostC matrix
  free(hostC);

  return 0;
}
