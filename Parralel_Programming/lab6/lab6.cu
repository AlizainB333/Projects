#include "hip/hip_runtime.h"
// MP Scan
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ...
// +
// lst[n-1]}

#include <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

__global__ void scan(float *input, float *output, int len) {
  //@@ Modify the body of this function to complete the functionality of
  //@@ the scan on the device
  //@@ You may need multiple kernel calls; write your kernels before this
  //@@ function and call them from the host

    // Memory for scan kernel 2*size
    __shared__ float scan_mem[2*BLOCK_SIZE];

    // Stride Index
    int idx = 2 * blockIdx.x * BLOCK_SIZE + threadIdx.x;
    

    // Load data into shared memory
    if (idx < len) {
      scan_mem[threadIdx.x] = input[idx];
    } else {
      scan_mem[threadIdx.x] = 0;
    }
    if (idx + BLOCK_SIZE < len) {
      scan_mem[threadIdx.x + BLOCK_SIZE] = input[idx + BLOCK_SIZE];
    } else {
      scan_mem[threadIdx.x + BLOCK_SIZE] = 0;
    }

    // Up-sweep / reduction step
    int stride = 1;
    while(stride < 2*BLOCK_SIZE) {
      __syncthreads();
      int index = (threadIdx.x + 1) * stride*2 - 1;

      if(index < 2*BLOCK_SIZE && (index-stride) >= 0) {
        scan_mem[index] += scan_mem[index-stride];
      }

      stride = stride*2;
    }

    // Post Scan Step 
    int stride_post = BLOCK_SIZE/2;
    while(stride_post > 0) {
      __syncthreads();
      int index = (threadIdx.x+1)*stride_post*2 - 1;

      if ((index+stride_post) < 2*BLOCK_SIZE) {
        scan_mem[index+stride_post] += scan_mem[index];
      }
      
      stride_post = stride_post / 2;
    }

    __syncthreads();

    if (idx < len) {
      output[idx] = scan_mem[threadIdx.x];
    }
    if (idx + blockDim.x < len) {
      output[idx + blockDim.x] = scan_mem[threadIdx.x + blockDim.x];
    } 
    
}

// Kernel 2: Store block sums into auxiliary array
__global__ void scanBlockSums(float *output, float *input, int len) {
  int scanBlockIdx = blockIdx.x;
  input[scanBlockIdx] = output[(scanBlockIdx) * blockDim.x * 2 + (blockDim.x*2 - 1)]; 
}

// Kernel 3: Add block sum to all values in the next block
__global__ void scanAddBlock(float *output, float *input, int len) {

  int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;
  if(idx < len && blockIdx.x > 0) {
    output[idx] += input[blockIdx.x-1];

    if (idx + blockDim.x < len) {
      output[idx + blockDim.x] += input[blockIdx.x-1];
    }
  }

}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *deviceInput;
  float *deviceOutput;
  int numElements; // number of elements in the list

  float *auxBlockDevice;

  args = wbArg_read(argc, argv);

  // Import data and create memory on host
  // The number of input elements in the input is numElements
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &numElements);
  hostOutput = (float *)malloc(numElements * sizeof(float));

  // Allocate GPU memory.
  wbCheck(hipMalloc((void **)&deviceInput, numElements * sizeof(float)));
  int gridSize = ceil((numElements *1.0) / (2*BLOCK_SIZE));

  wbCheck(hipMalloc((void **)&deviceOutput, numElements * sizeof(float)));
  wbCheck(hipMalloc((void **)&auxBlockDevice, gridSize * sizeof(float)));

  // Clear output memory.
  wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));

  // Copy input memory to the GPU.
  wbCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(float),
                     hipMemcpyHostToDevice));

  //@@ Initialize the grid and block dimensions here
  // int gridSize = ceil((numElements *1.0) / (2*BLOCK_SIZE));

  dim3 dimGrid(gridSize);
  dim3 dimBlock(BLOCK_SIZE);

  //@@ Modify this to complete the functionality of the scan
  //@@ on the deivce
  scan<<<dimGrid,dimBlock>>>(deviceInput,deviceOutput,numElements);
  hipDeviceSynchronize();

  scanBlockSums<<<dimGrid, BLOCK_SIZE>>>(deviceOutput, auxBlockDevice, numElements);
  hipDeviceSynchronize();

  scan<<<1, gridSize>>>(auxBlockDevice, auxBlockDevice, gridSize);
  hipDeviceSynchronize();

  scanAddBlock<<<dimGrid, dimBlock>>>(deviceOutput, auxBlockDevice, numElements);

  // Copying output memory to the CPU
  wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float),
                     hipMemcpyDeviceToHost));

  //@@  Free GPU Memory
  hipFree(deviceInput);
  hipFree(deviceOutput);

  wbSolution(args, hostOutput, numElements);

  free(hostInput);
  free(hostOutput);

  return 0;
}

